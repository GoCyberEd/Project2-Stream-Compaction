#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {

    	int * dev_data;

        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int pow, int pow1, int data[]) {
        	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
        	if (idx >= n) {
        		return;
        	}

        	int i = idx * pow1;
        	if (i < n) {
        		data[i + pow1 - 1] += data[i + pow - 1];
        	}
        }

        __global__ void kernDownSweep(int n, int pow, int pow1, int data[]) {
        	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
        	if (idx >= n) { return; }

        	int i = idx * pow1;
        	if (i < n) {
        		// Swap and sum
        		int aux = data[i + pow - 1];
        		data[i + pow - 1] = data[i + pow1 - 1];
        		data[i + pow1 - 1] += aux;
        	}
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
        	int ceil = ilog2ceil(n);
        	int ceilN = 1 << ceil;

        	hipMalloc((void**) &dev_data, ceilN * sizeof(int));
        	checkCUDAError("malloc dev_data failed");
        	hipMemset(dev_data, 0, n * sizeof(int));
        	checkCUDAError("hipMemset to clear array failed");

        	hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
        	checkCUDAError("hipMemcpy input host to device failed");
        	//hipMemcpy(dev_odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToDevice);

        	int pow, pow1, blocksPerGrid;

            timer().startGpuTimer();
            // Need two separate kernels, one for upsweep and one for down to ensure everything stays in sync
            // Can we just use sync_threads? No, becaue potentially multiple blocks
            // 1. upsweep (note it updates in place, hopefully this is okay? Just summing)
            // 2. Reset end of array to 0
            // 3. Downsweep
            for (int d = 0; d < ceil; d++) {
            	//pow = std::pow(2, d);
            	//pow1 = std::pow(2, d + 1);
            	pow = 1 << d;
            	pow1 = 1 << (d + 1);
            	blocksPerGrid = (ceilN / pow1 + BLOCK_SIZE - 1) / BLOCK_SIZE;
            	kernUpSweep<<< blocksPerGrid, BLOCK_SIZE >>>(ceilN, pow, pow1, dev_data);
            	checkCUDAError("kernUpSweep failed");
            }

            // Reset last value
            //int z = 0;
            //hipMemcpy(dev_data + ceilN - 1, &z, sizeof(int), hipMemcpyHostToDevice);
            hipMemset(dev_data + ceilN - 1, 0, sizeof(int));
            checkCUDAError("hipMemcpy zero failed");
            //dev_data[ceilN - 1] = 0;

            //for (int d = 0; d < ceil; d++) { start at end instead
            for (int d = ceil - 1; d >= 0; d--){
            	pow = 1 << d;
            	pow1 = 1 << (d + 1);
            	blocksPerGrid = (ceilN / pow1 + BLOCK_SIZE - 1) / BLOCK_SIZE;
            	kernDownSweep<<< blocksPerGrid, BLOCK_SIZE >>>(ceilN, pow, pow1, dev_data);
            	checkCUDAError("kernDownSweep failed");
            }
            timer().endGpuTimer();

            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("memcpy answer to host failed");

            hipFree(dev_data);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int * dev_bools;
            int * dev_indices;
            int * dev_scatter;
            int * dev_input;
            hipMalloc((void**) &dev_bools, n * sizeof(int));
            hipMalloc((void**) &dev_indices, n * sizeof(int));
            hipMalloc((void**) &dev_scatter, n * sizeof(int));
            hipMalloc((void**) &dev_input, n * sizeof(int));
            hipMemcpy(dev_input, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int host_indices[n];
            int host_bools[n];

        	int blocksPerGrid = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

        	// 1. Create boolean array
            Common::kernMapToBoolean<<< blocksPerGrid, BLOCK_SIZE  >>>(n, dev_bools, dev_input);
            hipMemcpy(host_bools, dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);
            // 2. Scan to generate indices
            scan(n, host_indices, host_bools);
            hipMemcpy(dev_indices, host_indices, n * sizeof(int), hipMemcpyHostToDevice);
            // 3. Scatter
            timer().startGpuTimer();
            Common::kernScatter<<< blocksPerGrid, BLOCK_SIZE >>>(n, dev_scatter, dev_input, dev_bools, dev_indices);
            timer().endGpuTimer();

            // Copy to output
            hipMemcpy(odata, dev_scatter, n * sizeof(int), hipMemcpyDeviceToHost);

            // Memory cleanup
            hipFree(dev_bools);
            hipFree(dev_indices);
            hipFree(dev_scatter);
            hipFree(dev_input);

            // Beware! Since exclusive scan, we won't count last element in
            // indices, let's fix that
            if (host_bools[n - 1] != 0) {
            	return host_indices[n - 1] + 1;
            } else {
            	return host_indices[n-1];
            }
        }
    }
}
